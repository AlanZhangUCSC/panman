#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <stack>
#include "panmanUtils.cuh"
#include "fitchSankoff.cuh"
#ifndef UTILS
#include "utils.hpp"
#endif
panmanUtils::Node::Node(std::string id, float len) {
    identifier = id;
    level = 1;
    branchLength = len;
    parent = nullptr;
}

panmanUtils::Node::Node(std::string id, Node* par, float len) {
    identifier = id;
    branchLength = len;
    parent = par;
    level = par->level + 1;
    par->children.push_back(this);
}

void stringSplit (std::string const& s, char delim, std::vector<std::string>& words) {
    size_t start_pos = 0, end_pos = 0, temp_pos = 0;
    while ((end_pos = s.find(delim, start_pos)) != std::string::npos) {
        if (end_pos >= s.length()) {
            break;
        }
        std::string sub;
        if (temp_pos == 0) {
            sub = s.substr(start_pos, end_pos-start_pos);
            if (std::count(sub.begin(), sub.end(), '\'') % 2 == 1) {
                temp_pos = start_pos;
            }
            else {
                words.emplace_back(sub);
            }
        }
        else {
            sub = s.substr(temp_pos, end_pos-temp_pos);
            if (std::count(sub.begin(), sub.end(), '\'') % 2 == 0) {
                temp_pos = 0;
                words.emplace_back(sub);
            }
        }
        // words.emplace_back(s.substr(start_pos, end_pos-start_pos));
        start_pos = end_pos+1;
    }
    auto last = s.substr(start_pos, s.size()-start_pos);
    if (last != "") {
        words.push_back(std::move(last));
    }
}


std::string stripString(std::string s){
    while(s.length() && s[s.length() - 1] == ' '){
        s.pop_back();
    }
    for(size_t i = 0; i < s.length(); i++){
        if(s[i] != ' '){
            return s.substr(i);
        }
    }
    return s;
}

panmanUtils::Node* panmanUtils::Tree::createTreeFromNewickString(std::string newickString) {
    newickString = stripString(newickString);

    Node* treeRoot = nullptr;

    std::vector<std::string> leaves;
    std::vector<size_t> numOpen;
    std::vector<size_t> numClose;
    std::vector<std::queue<float>> branchLen (128);  // will be resized later if needed
    size_t level = 0;
    // std::cout << newickString << std::endl;

    std::vector<std::string> s1;
    stringSplit(newickString, ',', s1);

    numOpen.reserve(s1.size());
    numClose.reserve(s1.size());
    

    for (auto s: s1) {
        size_t no = 0;
        size_t nc = 0;
        size_t leafDepth = 0;

        bool stop = false;
        bool branchStart = false;
        bool nameZone = false;
        bool hasApo = false;
        std::string leaf = "";
        std::string branch = "";

        for (auto c: s) {
            if (nameZone) {
                leaf += c;
                if (c == '\'') nameZone = false;
            } else if (c == '\'' && !nameZone) {
                nameZone = true;
                hasApo = true;
                leaf += c;
            } else if (c == ':') {
                stop = true;
                branch = "";
                branchStart = true;
            } else if (c == '(') {
                no++;
                level++;
                if (branchLen.size() <= level) {
                    branchLen.resize(level*2);
                }
            } else if (c == ')') {
                stop = true;
                nc++;
                // float len = (branch.size() > 0) ? std::stof(branch) : -1.0;
                float len = (branch.size() > 0) ? std::stof(branch) : 1.0;
                if (len == 0) len = 1.0;
                branchLen[level].push(len);
                level--;
                branchStart = false;
            } else if (!stop) {
                leaf += c;
                branchStart = false;
                leafDepth = level;

            } else if (branchStart) {
                if (isdigit(c)  || c == '.') {
                    branch += c;
                }
            }
        }
        if (hasApo && leaf[0] == '\'' && leaf[leaf.length()-1] == '\'') leaf = leaf.substr(1, leaf.length()-2);
        leaves.push_back(std::move(leaf));
        numOpen.push_back(no);
        numClose.push_back(nc);
        float len = (branch.size() > 0) ? std::stof(branch) : 1.0;
        if (len == 0) len = 1.0;
        branchLen[level].push(len);

        // Adjusting max and mean depths
        m_maxDepth = std::max(m_maxDepth, leafDepth);
        m_meanDepth += leafDepth;

    }


    m_meanDepth /= leaves.size();

    // std::cout << m_meanDepth << " " << level << std::endl;
    if (level != 0) {
        fprintf(stderr, "ERROR: incorrect Newick format!\n");
        exit(1);
    }

    m_numLeaves = leaves.size();

    std::stack<Node*> parentStack;
    int cc = 0;
    for (size_t i=0; i<leaves.size(); i++) {
        auto leaf = leaves[i];
        auto no = numOpen[i];
        auto nc = numClose[i];
        for (size_t j=0; j<no; j++) {
            std::string nid = newInternalNodeId();
            Node* newNode = nullptr;
            if (parentStack.size() == 0) {
                newNode = new Node(nid, branchLen[level].front());
                treeRoot = newNode;
            } else {
                newNode = new Node(nid, parentStack.top(), branchLen[level].front());
        
            }
            branchLen[level].pop();
            level++;

            if (allNodes.find(nid) != allNodes.end()) {
                fprintf(stderr, "ERROR: Node with id %s already exists!\n", nid.c_str());
            }
            allNodes[nid] = newNode;
            parentStack.push(newNode);
            cc++;
        }
        if (allNodes.find(leaf) != allNodes.end()) {
            fprintf(stderr, "ERROR: Node with id %s already exists!\n", leaf.c_str());
        }
        Node* leafNode = new Node(leaf, parentStack.top(), branchLen[level].front());
        allNodes[leaf] = leafNode;

        branchLen[level].pop();
        for (size_t j=0; j<nc; j++) {
            parentStack.pop();
            level--;
        }
    }

    if (treeRoot == nullptr) {
        fprintf(stderr, "WARNING: Tree found empty!\n");
    }

    treeRoot->branchLength = 0.0;
    std::cout << "Tree created with " << m_numLeaves << " leaves and " << allNodes.size() << " nodes\n";
    return treeRoot;
}


panmanUtils::Tree::Tree(std::ifstream& newick_ifstream) {
    std::string newickString;
    std::getline(newick_ifstream, newickString);
    root = createTreeFromNewickString(newickString);
}

int main(int argc, char**argv){
    std::ifstream newick_file_istream (argv[1]); // newick file
    std::string fname = argv[2]; // MSA fasta file
    std::string ref_file = argv[3]; // ref alignment fasta file

    auto start = std::chrono::high_resolution_clock::now();
    
    panmanUtils::Tree* T = new panmanUtils::Tree(newick_file_istream);
if (1) {
    printf("=== Tree information ===\n");
    printf("Number of nodes: %d\n", T->allNodes.size());
    printf("============================\n");
}
    std::unordered_map<std::string, std::string> seqs;
    std::unordered_map<std::string, std::string> refs;
    if(ref_file != "")
        read_seqs(ref_file, refs);

    utility::util *u = new utility::util(10000, 300, 0, 0);
    u->seq_file_name = fname;

    for (auto &a: refs)
    {
        u->ref_name = a.first;
        u->ref_seq = a.second;
        u->msa_len = a.second.size();
        u->consensus = a.second;
    }

    fitch_sankoff_on_gpu(T, seqs, u);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::nanoseconds total = end - start;
    printf("Total time: %lf mins\n", ((double)total.count()/1000000000)/60);
    return 0;
}
