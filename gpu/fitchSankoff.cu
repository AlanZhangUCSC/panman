#include "hip/hip_runtime.h"
#include "fitchSankoff.cuh"

char getNucleotideFromCode(int code) {
    switch(code) {
    case 1:
        return 'A';
    case 2:
        return 'C';
    case 4:
        return 'G';
    case 8:
        return 'T';
    case 5:
        return 'R';
    case 10:
        return 'Y';
    case 6:
        return 'S';
    case 9:
        return 'W';
    case 12:
        return 'K';
    case 3:
        return 'M';
    case 14:
        return 'B';
    case 13:
        return 'D';
    case 11:
        return 'H';
    case 7:
        return 'V';
    case 15:
        return 'N';
    default:
        return '-';
    }
}

void post_order_traversal(panmanUtils::Node* node, std::unordered_map<std::string, std::pair<int,int>>& node_id_map, int& id, int &leaf_id, int* leaf_or_not, int* child_map, int* order, int &max_order, std::unordered_map<int, std::string>& reverse_node_id_map, std::unordered_map<int, int>& internal_node_id_map, int& internal_node) {
    std::string node_name;
    int node_id;

    if (node->children.size() == 0) {
        node_name = node->identifier;
        node_id = id++;
        reverse_node_id_map[node_id] = node_name;
        node_id_map[node_name] = std::make_pair(node_id,leaf_id++);
        order[node_id] = 0;
        return;
    }
    
    for (auto child : node->children)
        post_order_traversal(child, node_id_map, id, leaf_id, leaf_or_not, child_map, order, max_order, reverse_node_id_map, internal_node_id_map, internal_node);
    

    node_name = node->identifier;
    node_id = id++;
    int internal_node_id = internal_node++;
    internal_node_id_map[node_id] = internal_node_id;
    reverse_node_id_map[node_id] = node_name;
    node_id_map[node_name] = std::make_pair(node_id, -1);
    bool first_child = false;
    int last_child_id = -1;
    int new_order = 0;
    for (auto child : node->children){
        std::string child_name = child->identifier;
        if (!first_child){
            first_child = true;
            leaf_or_not[node_id] = node_id_map[child_name].first;
        } else {
            child_map[last_child_id] = node_id_map[child_name].first;
        }
        last_child_id = node_id_map[child_name].first;
        if (new_order < order[last_child_id])
            new_order = order[last_child_id];
    }
    order[node_id] = new_order+1;
    if (new_order+1>max_order)
        max_order = new_order+1;

    return;
}

__device__
int char2int(char c)
{
    int s = 0;
    switch (c) {
        case 'A':
        return 1;
    case 'C':
        return 2;
    case 'G':
        return 4;
    case 'T':
        return 8;
    case 'R':
        return 5;
    case 'Y':
        return 10;
    case 'S':
        return 6;
    case 'W':
        return 9;
    case 'K':
        return 12;
    case 'M':
        return 3;
    case 'B':
        return 14;
    case 'D':
        return 13;
    case 'H':
        return 11;
    case 'V':
        return 7;
    case 'N':
        return 15;
    default:
        return 0;

    }
    return s;
}

__global__ 
void fs_fwd(
    char * d_seqs, 
    int * d_leaf_map, 
    int * d_leaf_or_not, 
    int * d_child_map, 
    int * d_order,
    int num_nodes,
    int num_leaves,
    int sites,
    int max_order,
    int32_t * d_ancestor[],
    int * d_internal_node_id_map)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    for (int i=bx;i<sites;i+=gs) //sites
    {
        int local_order = 0;
        // int ancestor_start_idx = 16*i*num_nodes;
        while(local_order <= max_order)
        {
            for (int j=tx; j<num_nodes; j+=bs) //nodes
            {
                int ancestor_start_idx2 = (d_internal_node_id_map[j])*16;
                int node_id = j;
                int order = d_order[node_id];
                if (order == local_order)
                {
                    int leaf_id = d_leaf_map[node_id];
                    int leaf_or_not = d_leaf_or_not[node_id];
                    // int v;
                    // if (leaf_or_not == -1) //leaf
                    // {
                    //     v = char2int(d_seqs[leaf_id*sites+i]);
                    //     for (int k=0;k<16;k++)
                    //     {
                    //         if (k==v)   d_ancestor[i][ancestor_start_idx2+k] = 0;
                    //         else        d_ancestor[i][ancestor_start_idx2+k] = INF;
                    //     }

                    // }
                    if (leaf_or_not>-1) //internal node
                    {
                        for (int k=0;k<16;k++)
                            d_ancestor[i][ancestor_start_idx2+k] = 0;
                        
                        bool found_min = false;
                        int child_id = leaf_or_not;
                        while (child_id != -1)
                        {
                            int child_start_idx = (d_internal_node_id_map[child_id])*16;
                            bool is_child_leaf = (d_leaf_or_not[child_id] == -1);
                            int leaf_v = -1;
                            if (is_child_leaf) //leaf
                                leaf_v=char2int(d_seqs[d_leaf_map[child_id]*sites+i]);
                            
                            for (int k=0;k<16;k++)
                            {
                                int min_value=INF;
                                if (is_child_leaf) 
                                    min_value = (k!=leaf_v);
                                else
                                {
                                    for (int l=0; l<16;l++) 
                                    {
                                        if (min_value > d_ancestor[i][child_start_idx+l]+(k!=l))
                                            min_value = d_ancestor[i][child_start_idx+l]+(k!=l);
                                    }
                                }
                                if (min_value < INF)
                                    d_ancestor[i][ancestor_start_idx2+k]+=min_value;
                                else 
                                    printf("Ideally should not happen.. Report to swalia@ucsd.edu, Thanks!! (Site: %d, Node: %d Child Node %d)\n", i, j, child_start_idx);
                            }
                            child_id = d_child_map[child_id];
                        }
                    }
                    // d_ancestor[ancestor_start_idx+node_id] = v;
                }
            }
            __syncthreads();
            local_order++;
        }
    }

}

__global__ 
void fs_bwd(
    int32_t *d_ancestor[],
    int8_t  *d_bwd_states,
    char * d_ref, 
    int * d_leaf_or_not, 
    int * d_child_map, 
    int * d_order,
    int num_nodes,
    int num_leaves,
    int sites,
    int max_order,
    char * d_seqs,
    int * d_leaf_map,
    int * d_internal_node_id_map)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    for (int i=bx;i<sites;i+=gs) //sites
    {
        int local_order = max_order;
        // int ancestor_start_idx = 16*i*num_nodes;
        int states_start_idx = i*num_nodes;
        while(local_order > -1)
        {
            for (int j=tx; j<num_nodes; j+=bs) //nodes
            {
                int node_id = j;
                int order = d_order[node_id];
                if (order == local_order)
                {
                    int leaf_or_not = d_leaf_or_not[node_id];
                    int v;
                    if (local_order == max_order) //root
                    {
                        v = char2int(d_ref[i]);
                        d_bwd_states[states_start_idx+j] = v;
                    }
                    if (leaf_or_not != -1) // internal nodes
                    {
                        v=d_bwd_states[states_start_idx+j];
                        int child_id = leaf_or_not;
                        while (child_id != -1)
                        {
                            int child_start_idx = (d_internal_node_id_map[child_id])*16;
                            int min_value = INF, min_ptr=-1;
                            
                            bool is_child_leaf = (d_leaf_or_not[child_id] == -1);
                            int leaf_v = -1;
                            if (is_child_leaf)
                                leaf_v = char2int(d_seqs[d_leaf_map[child_id]*sites+i]);
                            
                            for (int k=0;k<16;k++)
                            {
                                int value;
                                if (is_child_leaf) // leaf
                                    value = (k!=leaf_v);
                                else // internal node
                                    value = (k!=v)+d_ancestor[i][child_start_idx+k];
                                if (value<min_value)
                                {
                                    min_value = value;
                                    min_ptr = k;
                                }
                            }
                            d_bwd_states[states_start_idx+child_id] = min_ptr;
                            child_id = d_child_map[child_id];
                        }
                    }
                    // d_ancestor[ancestor_start_idx+node_id] = v;
                }
            }
            __syncthreads();
            local_order--;
        }
    }

}

__global__
void fs_assign_mut(
    int8_t * d_bwd_states,
    int * d_leaf_or_not, 
    int * d_child_map, 
    int * d_order,
    int num_nodes,
    int num_leaves,
    int sites,
    int max_order,
    int8_t * d_muts
){
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bs = blockDim.x;
    int gs = gridDim.x;

    int type = -1;

    for (int i=bx;i<sites;i+=gs) //sites
    {
        int local_order = max_order;
        int states_start_idx = i*num_nodes;
        int mut_start_idx = i*num_nodes;
        while(local_order > -1)
        {
            for (int j=tx; j<num_nodes; j+=bs) //nodes
            {
                int node_id = j;
                int order = d_order[node_id];
                if (order == local_order)
                {
                    int leaf_or_not = d_leaf_or_not[node_id];
                    if(leaf_or_not != -1) // internal nodes
                    {
                        int32_t node_state = d_bwd_states[states_start_idx + j]; 
                        if (node_state==-1)
                            d_muts[mut_start_idx+j]=-1;
                        else {
                            int child_id = leaf_or_not;
                            while(child_id > -1)
                            {
                                int32_t child_state = d_bwd_states[states_start_idx+child_id];
                                if (node_state != child_state) 
                                {
                                    if (node_state == 0) // insertion
                                        type = 2;
                                    else if (child_state == 0) // deletion
                                        type = 1;
                                    else // subs
                                        type = 0;
                                    d_muts[mut_start_idx+child_id] = (type<<4 | child_state);
                                } 
                                else 
                                {
                                    d_muts[mut_start_idx+child_id] = -1;
                                }
                                child_id = d_child_map[child_id];
                            }
                        }
                        
                    }
                }
            }
            __syncthreads();
            local_order--;
        }
    }
}

void allocate_mem_and_run(std::unordered_map<std::string, std::string>& seqs, int* leaf_or_not, int* child_map, int* order, std::unordered_map<std::string, std::pair<int, int>>& node_id_map,utility::util* u, std::unordered_map<int, std::string>&reverse_node_id_map, std::unordered_map<int, int>& internal_node_id_map){
    FILE *file = fopen("mutations.txt", "w");
    std::string error;
    
    int num_seq = u->num_tips;
    int * h_leaf_map = new int[u->num_nodes];
    for (int i=0; i<u->num_nodes; i++) 
        h_leaf_map[i]=-1;

    for (auto s: node_id_map) 
    {
        int leaf_id             = s.second.second;
        int node_id             = s.second.first;
        if (leaf_id>-1)
            h_leaf_map[node_id] = leaf_id;
    }

    if (u->ref_name == ""){
        printf("Currently the program requires users to pass reference to perform fitch-Sankoff\n");
        exit(0);
    }
    
    int *d_internal_node_id_map, *h_internal_node_id_map;
    char *d_seqs,    *h_seqs,    *h_seqs_global;
    char *d_ref_seq, *h_ref_seq, *h_ref_seq_global;
    int32_t *d_ancestor[u->local_batch_size];
    int8_t  *d_bwd_states;
    // [u->local_batch_size];
    int8_t  *d_muts;
    // [u->local_batch_size];
    int *d_leaf_or_not = new int [u->num_nodes];
    int *d_child_map   = new int [u->num_nodes];
    int *d_order       = new int [u->num_nodes];
    int *d_leaf_map    = new int [u->num_nodes];
    
    // allocate memory on CPU
    // h_seqs_global    = (char *)malloc(u->global_batch_size*num_seq*sizeof(char));
    // h_ref_seq_global = (char *)malloc(u->global_batch_size*num_seq*sizeof(char));
    h_seqs           = (char *)malloc(u->local_batch_size*num_seq*sizeof(char));
    h_ref_seq        = (char *)malloc(u->local_batch_size*sizeof(char));
    h_internal_node_id_map = (int*) malloc(u->num_nodes*sizeof(int));

    for (auto n: internal_node_id_map)
        h_internal_node_id_map[n.first] = n.second;
    
    // allocate memory on GPU
    size_t freeMemory, totalMemory;
    hipMemGetInfo(&freeMemory, &totalMemory);
    printf("============= Memory Usage on GPU ===============\n");
    printf("Total memory on GPU:%lf GB\n", (double)totalMemory/(1024*1024*1024));
    printf("Available memory: %lf GB\n", (double)freeMemory/(1024*1024*1024));
    
    printf("Allocating %lf GB for Seqs on GPU\n", (double)num_seq*u->local_batch_size/(1024*1024*1024));
    printf("Allocating %lf GB for Tree structure on GPU\n", (double)u->num_nodes*5*4/(1024*1024*1024));
    printf("Allocating %lf GB for Fitch forward states on GPU\n",((double)16*(u->num_nodes-u->num_tips)*u->local_batch_size*4 + u->local_batch_size * 8)/(1024*1024*1024));
    printf("Allocating %lf GB for Ref seq on GPU\n", (double)u->local_batch_size/(1024*1024*1024));
    printf("Allocating %lf GB for Fitch bwd states seq on GPU\n", (double)u->local_batch_size*u->num_nodes/(1024*1024*1024));
    printf("Allocating %lf GB for Fitch mutation seq on GPU\n", (double)u->num_nodes*u->local_batch_size/(1024*1024*1024)); 
    double total_usage = (double)num_seq*u->local_batch_size/(1024*1024*1024) + \
                         (double)u->num_nodes*5*4/(1024*1024*1024) + \
                         ((double)16*(u->num_nodes-u->num_tips)*u->local_batch_size*4 + u->local_batch_size *8)/(1024*1024*1024) + \
                         (double)u->local_batch_size/(1024*1024*1024) + \
                         (double)u->local_batch_size*u->num_nodes/(1024*1024*1024) + \
                         (double)u->num_nodes*u->local_batch_size/(1024*1024*1024);        

    printf("\nTotal Usage: %lf GB\n", total_usage);
    
    for (int i=0; i<u->local_batch_size; i++)
    {
        hipMalloc(&d_ancestor[i], 16*(u->num_nodes-u->num_tips)*sizeof(int32_t));
        // hipMalloc(&d_bwd_states[i], u->num_nodes*sizeof(int8_t));
        // hipMalloc(&d_muts[i], u->num_nodes*sizeof(int8_t));
    }
    
    int32_t** d_vectorsArray;
    int8_t** d_bwd_states_array;
    int8_t** d_muts_array;
    hipMalloc(&d_vectorsArray, u->local_batch_size * sizeof(int32_t*));
    // hipMalloc(&d_bwd_states_array, u->local_batch_size * sizeof(int8_t*));
    // hipMalloc(&d_muts_array, u->local_batch_size * sizeof(int8_t*));
    
    hipMemcpy(d_vectorsArray, d_ancestor, u->local_batch_size * sizeof(int32_t*), hipMemcpyHostToDevice);
    // hipMemcpy(d_bwd_states_array, d_bwd_states, u->local_batch_size * sizeof(int8_t*), hipMemcpyHostToDevice);
    // hipMemcpy(d_muts_array, d_muts, u->local_batch_size * sizeof(int8_t*), hipMemcpyHostToDevice);

    error = hipGetErrorString(hipGetLastError()); 
    if (error != "no error") 
    {
        printf("ERROR: hipMallocPitch %s!\n", error.c_str());
        exit(0);
    }
    hipMalloc(&d_seqs, num_seq*u->local_batch_size*sizeof(char));
    hipMalloc(&d_leaf_or_not, u->num_nodes*sizeof(int));
    hipMalloc(&d_child_map, u->num_nodes*sizeof(int));
    hipMalloc(&d_order, u->num_nodes*sizeof(int));
    hipMalloc(&d_leaf_map, u->num_nodes*sizeof(int));
    hipMalloc(&d_ref_seq, u->local_batch_size*sizeof(char));
    hipMalloc(&d_bwd_states, u->local_batch_size*u->num_nodes*sizeof(int8_t));
    hipMalloc(&d_muts, u->num_nodes*u->local_batch_size*sizeof(int8_t));
    hipMalloc(&d_internal_node_id_map, u->num_nodes*sizeof(int));

    hipMemGetInfo(&freeMemory, &totalMemory);
    printf("Free memory after allocation: %lf GB\n", (double)freeMemory/(1024*1024*1024));
    printf("=================================================\n");

    // Requires one time transfer
    hipMemcpy(d_leaf_or_not, leaf_or_not, u->num_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_child_map, child_map, u->num_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_order, order, u->num_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_leaf_map, h_leaf_map, u->num_nodes*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_internal_node_id_map, h_internal_node_id_map, u->num_nodes*sizeof(int), hipMemcpyHostToDevice);
    
    error = hipGetErrorString(hipGetLastError()); 
    if (error != "no error") printf("ERROR: Cuda memcpy Trees structure %s!\n", error.c_str());
    

    size_t fitch_global_position = 0;
    while (fitch_global_position<u->ref_seq.size())
    {
        auto batch_start = std::chrono::high_resolution_clock::now();
        if (u->ref_seq.size()-fitch_global_position<u->global_batch_size)
            u->global_batch_size = u->ref_seq.size()-fitch_global_position;
        fprintf(stderr,"Handling data from position %ld to %d\n", fitch_global_position, fitch_global_position+u->global_batch_size);
        
        std::unordered_map<std::string, std::string> mutations;
        // Reading Sequences from Disk
        auto read_start = std::chrono::high_resolution_clock::now();
        read_seqs(u->seq_file_name, seqs, fitch_global_position, u->global_batch_size, 1);
        auto read_end = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds read_time = read_end - read_start;
        fprintf(stderr, "Reading Sequence took %lf mins\n", ((double)read_time.count()/1000000000)/60);
    
        size_t fitch_local_position = 0;
        while (fitch_local_position<u->global_batch_size)
        {
            if (fitch_local_position+u->local_batch_size>u->global_batch_size)
                u->local_batch_size = u->global_batch_size-fitch_local_position;
            fprintf(stderr, "%d...", fitch_local_position);
            for (auto s: node_id_map) {
                int leaf_id             = s.second.second;
                int node_id             = s.second.first;
                std::string node_name   = s.first;
                if (leaf_id>-1){
                    std::string seq = seqs[node_name];
                    for (int i=0;i<u->local_batch_size;i++)
                        h_seqs[leaf_id*u->local_batch_size+i]=seq[fitch_local_position+i];
                }
            }

            for (int i=0; i<u->local_batch_size;i++)
                h_ref_seq[i]=u->ref_seq[fitch_global_position+fitch_local_position+i];

            hipMemcpy(d_seqs, h_seqs, num_seq*u->local_batch_size*sizeof(char), hipMemcpyHostToDevice);
            hipMemcpy(d_ref_seq, h_ref_seq, u->local_batch_size*sizeof(char), hipMemcpyHostToDevice);
            
            error = hipGetErrorString(hipGetLastError()); 
            if (error != "no error") printf("ERROR: Cuda memcpy %s!\n", error.c_str());

            auto fwd_start = std::chrono::high_resolution_clock::now();
            fs_fwd<<<1024,1024>>>(d_seqs, d_leaf_map, d_leaf_or_not, d_child_map, d_order, u->num_nodes, seqs.size(), u->local_batch_size, u->max_order, d_vectorsArray, d_internal_node_id_map);
            hipDeviceSynchronize();
            auto fwd_end = std::chrono::high_resolution_clock::now();
            std::chrono::nanoseconds fwd_time = fwd_end - fwd_start;
            error = hipGetErrorString(hipGetLastError());
            if (error != "no error")
                printf("ERROR: After fs_fwd - %s!\n", error.c_str());

if (0)
{
    int16_t * h_ancestor = new int16_t[u->local_batch_size*u->num_nodes*16];    
    hipMemcpy(h_ancestor,    d_ancestor,    (u->local_batch_size*u->num_nodes*16)*sizeof(int16_t), hipMemcpyDeviceToHost);
    error = hipGetErrorString(hipGetLastError());
    if (error != "no error")
        printf("ERROR: After Copy to Host %s!\n", error.c_str());
    for(auto n:node_id_map){
        int node_id = n.second.first;
        printf("%s\n",n.first.c_str());
        for (int j=0;j<16;j++){
            printf("%d\t", h_ancestor[node_id*16+j]);
        }
        printf("\n");
    }
}
        

            auto bwd_start = std::chrono::high_resolution_clock::now();
            fs_bwd<<<1024,1024>>>(d_vectorsArray, d_bwd_states, d_ref_seq, d_leaf_or_not, d_child_map, d_order, u->num_nodes, seqs.size(), u->local_batch_size, u->max_order, d_seqs, d_leaf_map, d_internal_node_id_map);
            hipDeviceSynchronize();
            auto bwd_end = std::chrono::high_resolution_clock::now();
            std::chrono::nanoseconds bwd_time = bwd_end-bwd_start;

            error = hipGetErrorString(hipGetLastError());
            if (error != "no error")
                printf("ERROR: After fs_bwd - %s!\n", error.c_str());

if (0)
{
    int16_t * h_states = new int16_t[u->local_batch_size*u->num_nodes];    
    hipMemcpy(h_states,    d_bwd_states,    (u->local_batch_size*u->num_nodes)*sizeof(int16_t), hipMemcpyDeviceToHost);
    error = hipGetErrorString(hipGetLastError());
    if (error != "no error")
        printf("ERROR: After Copy to Host %s!\n", error.c_str());
    int position = 0;
    for(auto n:node_id_map){
        int node_id = n.second.first;
        int16_t v = h_states[node_id + position*u->num_nodes];
        printf("%d %s\n",node_id, n.first.c_str());
        printf("%d\t",v);
        
        printf("\n");
    }
}
           

            auto mut_start = std::chrono::high_resolution_clock::now();
            fs_assign_mut<<<1024,1024>>>(d_bwd_states, d_leaf_or_not, d_child_map, d_order, u->num_nodes, seqs.size(), u->local_batch_size, u->max_order, d_muts);
            hipDeviceSynchronize();
            auto mut_end = std::chrono::high_resolution_clock::now();
            std::chrono::nanoseconds mut_time = mut_end-mut_start;
            error = hipGetErrorString(hipGetLastError());
            if (error != "no error")
                printf("ERROR: After fs_assign_mut - %s!\n", error.c_str());

if(1)
{

    int8_t * h_muts = new int8_t[u->num_nodes*u->local_batch_size];
    hipMemcpy(h_muts,    d_muts,    (u->num_nodes*u->local_batch_size)*sizeof(int8_t), hipMemcpyDeviceToHost);

    error = hipGetErrorString(hipGetLastError());
    if (error != "no error")
        printf("ERROR: After Copy to Host %s!\n", error.c_str());
    for(auto n:node_id_map){
        int node_id = n.second.first;
        if (reverse_node_id_map[node_id] == "node_1")
            continue;
        std::string s = "";
        
        // s+= reverse_node_id_map[node_id];
        // s += ":\t";
        int count_mutations=0;
        for (int i=0; i<u->local_batch_size; i++)
        {
            int8_t v = h_muts[i*u->num_nodes + node_id];
            if (v!=-1)
            {
                count_mutations++;
                int type = v>>4;
                int8_t c = v & 0x0F;
                
                if (type == 0) s+= 'S';
                if (type == 1) s+= 'D';
                if (type == 2) s+= 'I';
                s += std::to_string(fitch_global_position+fitch_local_position+i);
                s += getNucleotideFromCode(c);
                s += '\t';
            }
        }
        // s += "\n";
        if (count_mutations>0)
        {
            // fprintf(file, "%s",s.c_str());
            if (mutations.find(reverse_node_id_map[node_id]) == mutations.end())
                mutations[reverse_node_id_map[node_id]]="";
            mutations[reverse_node_id_map[node_id]].append(s);
        }
    }
}
            fitch_local_position += u->local_batch_size;
        }
        fprintf(stderr,"%d...\n", fitch_local_position);
        auto batch_end = std::chrono::high_resolution_clock::now();
        std::chrono::nanoseconds batch_time = batch_end - batch_start;
        fprintf(stderr,"Batch completed in %lf mins\n", ((double)batch_time.count()/1000000000)/60);

        for (int i=0; i<u->global_batch_size; i++)
        {
            char c = u->ref_seq[fitch_global_position+i];
            if (c=='-')
            {
                bool found = false;
                for (auto s: seqs) {
                    if (s.second[i] != '-')
                    {
                        u->consensus[fitch_global_position+i] = s.second[i];
                        found = true;
                        break;
                    }
                }
                if (found == false) 
                {
                    printf("Error: Position %d has all -\n", i);
                    exit(0);
                }
            }
        }

if (1)
{
    for (auto &s: mutations)
    {
        fprintf(file, "%s:\t", s.first.c_str());
        fprintf(file, "%s\n", s.second.c_str());
    }
}

        fitch_global_position += u->global_batch_size;
    }
    
if (1){
    // node_1 mutations
    std::string s = "node_1:\t";
    for (int i=0; i<u->ref_seq.size(); i++)
    {
        char child_state = u->ref_seq[i];
        char node_state = u->consensus[i];
        if (node_state != child_state) 
        {
            if (node_state == '-') // insertion
                s += 'I';
            else if (child_state == '-') // deletion
                s += 'D';
            else // subs
                s += 'S';
            s+=std::to_string(i);
            s+=child_state;
            s+='\t';
        } 
    }
    s += "\n";
    fprintf(file, "%s",s.c_str());
}
    fclose(file);

    file = fopen("consensus.txt", "w"); 
    fprintf(file, "%s",u->consensus.c_str());
    fclose(file);

    hipFree(d_seqs);
    hipFree(d_ref_seq);
    hipFree(d_bwd_states);
    hipFree(d_ancestor);
    hipFree(d_vectorsArray);
    hipFree(d_leaf_or_not);
    hipFree(d_child_map);
    hipFree(d_order);
    hipFree(d_leaf_map);
    hipFree(d_muts);
    // hipFree(d_muts_array);
    // hipFree(d_bwd_states_array);

    return;
}

void fitch_sankoff_on_gpu(panmanUtils::Tree* T, std::unordered_map<std::string, std::string>& seqs, utility::util* u) {
    printf("Creating Tree for Device\n");
    int *leaf_or_not,*child_map, *order;
    leaf_or_not = (int*)malloc(T->allNodes.size()*sizeof(int));
    child_map = (int*)malloc(T->allNodes.size()*sizeof(int));
    order = (int*)malloc(T->allNodes.size()*sizeof(int));
    for (int i = 0; i < T->allNodes.size(); i++) {
        child_map[i] = -1;
        leaf_or_not[i] = -1;
        order[i] = 0;
    }

    // create a mapping between node's old ids and new ids
    std::unordered_map<std::string, std::pair<int,int>> node_id_map;
    std::unordered_map<int, int> internal_node_id_map; // ids for nodes except leaf nodes
    std::unordered_map<int, std::string> reverse_node_id_map;
    int id = 0, leaf_id=0, max_order=0; int internal_node=0;
    // node_id_map[T->root->identifier.c_str()] = std::make_pair(id,-1);
    // internal_node_id_map[id] = internal_node;
    post_order_traversal(T->root, node_id_map, id, leaf_id, leaf_or_not, child_map, order, max_order, reverse_node_id_map, internal_node_id_map, internal_node);
    u->max_order = max_order;
    u->num_nodes = id;
    u->num_tips = leaf_id;
    
if (0) {
    for (auto node: T->allNodes) {
        int new_id = node_id_map[node.first.c_str()].first;
        printf("%d: %s, is a leaf: %d and has children: %d, and order: %d\n", new_id, node.first.c_str(), leaf_or_not[new_id], child_map[new_id], order[new_id]);
        printf("Position %d - char %c\n", 0, seqs[node.first][0]);
    }
}  

    allocate_mem_and_run(seqs, leaf_or_not, child_map, order, node_id_map,u, reverse_node_id_map, internal_node_id_map);

}
